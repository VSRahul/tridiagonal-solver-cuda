#include "hip/hip_runtime.h"
/*******************************************************************************************************
                              University of Illinois/NCSA Open Source License
                                 Copyright (c) 2012 University of Illinois
                                          All rights reserved.

                                        Developed by: IMPACT Group
                                          University of Illinois
                                      http://impact.crhc.illinois.edu

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), 
to deal with the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense,
 and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

  Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimers.
  Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimers in the documentation and/or other materials provided with the distribution.
  Neither the names of IMPACT Group, University of Illinois, nor the names of its contributors may be used to endorse or promote products derived from this Software without specific prior written permission.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE
OR THE USE OR OTHER DEALINGS WITH THE SOFTWARE.

*******************************************************************************************************/


#include <stdio.h>
#include <complex.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "cusparse_ops.hxx"
#include <stddef.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <sys/sysinfo.h>
#include <helper_string.h>    // helper for string parsing
#include <hip/hip_runtime_api.h>      // helper for cuda error checking functions
#include "datablock.h"
#include <assert.h>

#define DEBUG 0
#define PI 3.141592653589793
static double get_second (void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
}

// findBestGrid
template <typename T, typename T_REAL>
int find_marshaled_index(Datablock<T, T_REAL> *data, int m)
{
    int bx;
    int by;
    int bdx = data->blockDim.x;
    int tx;
    int ty;
    int blockIndex;
    int h_stride = data->h_stride;
    int l_stride = data->b_dim;
    int blockOffset;
    int subBlockIndex;
    int subBlockOffset;
    int mNewIndex;

    blockIndex = m/(h_stride*l_stride);
    blockOffset = m%(h_stride*l_stride);
    subBlockIndex = blockOffset/(h_stride*bdx);
    subBlockOffset = blockOffset%(h_stride*bdx);
    bx = subBlockIndex;
    by = blockIndex;
    tx = ((subBlockOffset%h_stride)%bdx);
    ty = subBlockOffset/h_stride;
    mNewIndex = (by*l_stride*h_stride) + (tx*l_stride) + (bx*bdx) + ty + bdx*l_stride*((subBlockOffset%h_stride)/bdx);
    return mNewIndex;
}

template <typename T> 
void findBestGrid(int m, int tile_marshal, int *p_m_pad, int *p_b_dim, int *p_s, int *p_stride)
{
    int b_dim, m_pad, s, stride;
    int B_DIM_MAX, S_MAX;
    
    // due to shared memory being limited??
    if ( sizeof(T) == 4) 
    {
        B_DIM_MAX = 256;
        S_MAX     = 512;
    }
    else if (sizeof(T) == 8)
    { /* double and complex */
        B_DIM_MAX = 128;
        S_MAX     = 256;     
    }
    else 
    { /* doubleComplex */
        B_DIM_MAX = 64;
        S_MAX     = 128;    
    }
    
    /* b_dim must be multiple of 32 */
    // since warp size is 32?
    if ( m < B_DIM_MAX * tile_marshal ) 
    {
        b_dim = max(32, (m/(32*tile_marshal))*32);
        s = 1;
        m_pad = ((m + b_dim * tile_marshal - 1)/(b_dim * tile_marshal)) * (b_dim * tile_marshal);
        // m_pad is m increased to the closest multiple of (b_dim * tile_marshal)  
        stride = m_pad/(s*b_dim);    
    }
    else 
    {
        b_dim = B_DIM_MAX;
        
        s = 1;
        do {       
            int s_tmp = s * 2;
            int m_pad_tmp = ((m + s_tmp*b_dim*tile_marshal - 1)/(s_tmp*b_dim*tile_marshal)) * (s_tmp*b_dim*tile_marshal);           
            float diff = (float)(m_pad_tmp - m)/float(m);
            /* We do not want to have more than 20% oversize ... WHY?*/
            if ( diff < .2 ){
                s = s_tmp;      
            }
            else {
                break;
            }
        } while (s < S_MAX);

        m_pad = ((m + s*b_dim*tile_marshal -1)/(s*b_dim*tile_marshal)) * (s*b_dim*tile_marshal);        
        stride = m_pad/(s*b_dim);
        // m_pad = h_stride * l_stride * gridDim.y
    }
      
    *p_stride = stride;
    *p_m_pad  = m_pad;
    *p_s      = s;
    *p_b_dim  = b_dim;        
}

template <typename T, typename T_REAL> 
void tridiagonalSolver(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T* bNew, T *rhsUpdateArray, const int m);

template <typename T, typename T_REAL> 
void tridiagonalSolverHost(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T *bNew, T *rhsUpdateArray, T* x, const int m);


//template<typename T>
void setConstants(hipDoubleComplex *dx_2InvComplex);

//utility
#define EPS 1e-20

// mv_test fnxn takes tridiagonal matrix A (with diagonals a, b, c) and multiplies it with x (d) to give B (x)  
template <typename T> 
void mv_test
(
    T* x,               // result B
    const T* a,         // lower diagonal
    const T* b,         // diagonal
    const T* c,         // upper diagonal
    const T* d,         // variable vector
    const int len       // length of the matrix
)
{
    printf("Multiplying A with result x to get B ...\n");
    int m=len;
    x[0] =  cuAdd(  cuMul(b[0],d[0]), 
                        cuMul(c[0],d[1]));
    // does the multiplication of the first row
    
    // multiplication of rows 1 to m-1
    for(int i=1; i<m-1; i++)
    {   
        //x[i]=  a[i]*d[i-1]+b[i]*d[i]+c[i]*d[i+1];
        x[i]=  cuMul(a[i], d[i-1]);
        x[i]=  cuFma(b[i], d[i], x[i]);
        // cuFma first multiplies 1st 2 params and then adds 3rd one  
        x[i]=  cuFma(c[i], d[i+1], x[i]);
    }
        
    // multiplication of last row m
    x[m-1]= cuAdd(cuMul(a[m-1],d[m-2]) , cuMul(b[m-1],d[m-1]) );
    printf("Multiplication done.\n\n");
}

// mv_test fnxn takes tridiagonal matrix A (with diagonals a, b, c) and multiplies it with x (d) to give B (x)  
template <typename T> 
void mv_test_update
(
    T* x,               // result B
    T dx_2InvComplex,         // lower diagonal
    const T* b,         // diagonal
    const T* d,         // variable vector
    const int len       // length of the matrix
)
{
    printf("Multiplying updated A with result x to get new B ...\n");
    int m = len;
    x[0] =  cuAdd(cuMul(b[0], d[0]), 
                        cuMul(dx_2InvComplex, d[1]));
    // does the multiplication of the first row
    
    // multiplication of rows 1 to m-1
    for(int i=1; i<m-1; i++)
    {   
        //x[i]=  a[i]*d[i-1]+b[i]*d[i]+c[i]*d[i+1];
        x[i]=  cuMul(dx_2InvComplex, d[i-1]);
        x[i]=  cuFma(b[i], d[i], x[i]);
        // cuFma first multiplies 1st 2 params and then adds 3rd one  
        x[i]=  cuFma(dx_2InvComplex, d[i+1], x[i]);
    }
        
    // multiplication of last row m
    x[m-1]= cuAdd(cuMul(dx_2InvComplex, d[m-2]), cuMul(b[m-1], d[m-1]) );
    printf("Multiplication done.\n\n");
}


// compare_result<T, T_REAL>(h_b, h_b_back, 1, m, 1, 1e-10, 1e-10, 50, 3, b_dim);
template <typename T, typename T_REAL> 
void compare_result
(
    const T *x,             // B vector in Ax = B, given to us 
    const T *y,             // B vector in Ax = B, calc from GPU results 
    const int len,          // length of matrix 
    const T_REAL abs_err,   // for abs error checking
    const T_REAL re_err,    // for rel error checking
    const int p_bound,      // bound on error counting
    const int tx
)
{
    printf("Comparing computed B with given B.\n");
    T_REAL err = 0.0;
    T_REAL sum_err = 0.0;
    T_REAL total_sum = 0.0;
    T_REAL r_err = 1.0;
    T_REAL x_2 = 0.0;
    int p = 0; //error counter
    int t = 0; //check counter
    
    for(int i=0;i<len;i++)
    {
        T diff = cuSub(x[i], y[i]);
        err = cuReal(cuMul(diff, hipConj(diff) ));
        sum_err +=err;
        x_2 = cuReal(cuMul(x[i], hipConj(x[i])));
        total_sum += x_2;
        
        //avoid overflow in error check
        r_err = x_2 > EPS ? err/x_2:0.0;
        if(err > abs_err || r_err > re_err)
        {
            if(p < p_bound)
            {
                printf("Error occurred at element %2d, cpu = %E and gpu = %E at %d\n", i, cuReal(x[i]), cuReal(y[i]), i%tx);
                printf("Its absolute error is %le and relative error is %le.\n", err, r_err);
            }
            p++;
        }
        
        if(t < 16)
        {
            printf("Checked element %2d, cpu = %E and gpu = %E\n", i, cuReal(x[i]), cuReal(y[i]));
            t++;
        }
    }
    if(p == 0)
        printf("All correct.\n\n");
    else
        printf("There are %d errors.\n\n", p);

    printf("Total absolute error is %le\n",sqrt(sum_err));
    printf("Total relative error is %le\n",sqrt(sum_err)/sqrt(total_sum));
    printf("Comparing done.\n\n");
}

// This is a testing gtsv function
template <typename T, typename T_REAL> 
void gtsv_randomMatrix(int m, int steps)
{
    // each array is a set of elements in a diagonal stored in contiguous mem locations.
    T *h_dl;            // set of lower diagonal elements of mat A (n-1 elements)
    T *h_d;             // diagonal elements of mat A (n elements)
    T *h_du;            // set of upper diagonal elements of mat A (n-1 elements)
    T *h_b;             // RHS array has n elements
    T *h_field;         // field array to store x in cpu computation
    T *h_rhsUpdateArray;// array to be multiplied for RHS update
    T *h_bNew;          // bNew after n steps on CPU
    
    T *h_x_gpu;     // results from GPU
    T *h_bNew_gpu;  // copies updated RHS from GPU
    T *h_b_back;    // stores b computed from GPU results
    T *h_bNew_back; // stores updated RHS computed from GPU results
    T       *h_Ex;  // initial Gaussian wave
    T_REAL  *h_x;   // distance from origin
    T_REAL  *h_n;   // refractive index profile

    // vectors on the device
    T       *dl;    // lower diagonal in B
    T       *d;     // main diagonal in B
    T       *du;    // upper diagonal in B
    T       *b;     // B in Ax = B
    T       *bNew;  // to store new RHS array on device
    T       *rhsUpdateArray; // to store array which is to be multipled to get B new

    // constants
    // printf("-------------------------------------------\n");
    // printf("steps = %d\n", steps);    
    // printf("-------------------------------------------\n");
    T_REAL halfWidth= 2;
    T_REAL simDomain= 40;
    T_REAL dx       = simDomain/(m+2);
    T_REAL dx_2Inv  = 1/(dx*dx);
    T_REAL dz       = 0.55;
    T_REAL dzInv    = 1/dz;
    T_REAL nCore    = 1.5;
    T_REAL nClad    = 1.48;
    T_REAL nRef     = 1.48;
    T_REAL lambda   = 1.55;
    T_REAL k0       = 2*PI/lambda;
    T_REAL k0_2     = k0*k0;
    T_REAL beta     = k0*nRef;
    T dx_2InvComplex= cuGet<T>(-dx_2Inv, (T_REAL)0.0);
    hipDoubleComplex dx_2InvComplex_1= cuGet<hipDoubleComplex>(dx_2Inv, (T_REAL)0.0);

    // parameter declaration
    int s;                  // gridDim.x (or gridDim.y?)
    int stride;             // number of elements given to a thread
    int b_dim, m_pad;       // b_dim is used, for what? m_pad is the new size of the diagonal arrays after data transformation
    int tile_marshal = 16;  // blockDim in each direction for data marshaling
    int T_size = sizeof(T); // size of T

    // finds appropriate gridSize for data marshaling (will be referred to as DM from now on)
    findBestGrid<T>(m, tile_marshal, &m_pad, &b_dim, &s, &stride);
    printf("m = %d, m_pad = %d, s = %d, b_dim (l_stride) = %d, stride (h_stride) = %d\n", m, m_pad, s, b_dim, stride);    

    // int local_reduction_share_size   = 2*b_dim*3*T_size;
    // int global_share_size            = 2*s*3*T_size;
    // int local_solving_share_size     = (2*b_dim*2+2*b_dim+2)*T_size;
    // int marshaling_share_size        = tile_marshal*(tile_marshal+1)*T_size;
    
    Datablock<T, T_REAL> data(m, m_pad, s, steps, dx_2InvComplex, b_dim);
    dim3 gridDim(b_dim/tile_marshal, s);        // g_data
    dim3 blockDim(tile_marshal, tile_marshal);  // b_data
    data.setLaunchParameters(gridDim, blockDim, s, b_dim, tile_marshal, stride);

    // allocation of host vectors
    checkCudaErrors(hipHostMalloc((void **) &h_d, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_b, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_n, sizeof(T_REAL) * (m+2)));
    checkCudaErrors(hipHostMalloc((void **) &h_x, sizeof(T_REAL) * (m+2)));
    checkCudaErrors(hipHostMalloc((void **) &h_dl, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_du, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_field, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_Ex, T_size * (m+2)));
    checkCudaErrors(hipHostMalloc((void **) &h_x_gpu, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_b_back, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_bNew, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_bNew_gpu, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_bNew_back, T_size * m));
    checkCudaErrors(hipHostMalloc((void **) &h_rhsUpdateArray, T_size * m));
    // file is meant to store result at every step
    FILE *fp1   = fopen("output", "w");

    // setting refractive index profile, distance and initial source conditions
    int i;
    for(i=0; i<m+2; i++)
    {
        h_x[i]  = -20 + i*dx;
        if(h_x[i] > -halfWidth && h_x[i] < halfWidth)
            h_n[i] = nCore;
        else
            h_n[i] = nClad;
        h_Ex[i] = cuGet<T>(exp(-h_x[i]*h_x[i]/16), (T_REAL)0.0);
    }
    
    // allocation of device vectors
    checkCudaErrors(hipMalloc((void **)&dl,    T_size*m)); 
    checkCudaErrors(hipMalloc((void **)&du,    T_size*m)); 
    checkCudaErrors(hipMalloc((void **)&d,     T_size*m)); 
    checkCudaErrors(hipMalloc((void **)&b,     T_size*m));
    checkCudaErrors(hipMalloc((void **)&bNew,  T_size*m));
    checkCudaErrors(hipMalloc((void **)&rhsUpdateArray,  T_size*m));

    // the device vectors corresponding to entries of tridiagonal matrix are all set to zero
    checkCudaErrors(hipMemset(d,  0, m * T_size));
    checkCudaErrors(hipMemset(dl, 0, m * T_size));
    checkCudaErrors(hipMemset(du, 0, m * T_size));
    checkCudaErrors(hipMemset(b,  0, m * T_size));
    
    // T gammaLeft     = cuDiv(h_Ex[1], h_Ex[2]);
    // T gammaRight    = cuDiv(h_Ex[m], h_Ex[m-1]);
    T gammaLeft     = cuGet<T>((T_REAL)0.0, (T_REAL)0.0);
    T gammaRight    = cuGet<T>((T_REAL)0.0, (T_REAL)0.0);
    T constLhsTop   = cuGet<T>(2*dx_2Inv - k0_2*(pow(h_n[1], 2) - pow(nRef, 2)), 4*beta*dzInv);
    T constLhsBot   = cuGet<T>(2*dx_2Inv - k0_2*(pow(h_n[m], 2) - pow(nRef, 2)), 4*beta*dzInv);
    T constRhsTop   = cuGet<T>(-2*dx_2Inv + k0_2*(pow(h_n[1], 2) - pow(nRef, 2)), 4*beta*dzInv);
    T constRhsBot   = cuGet<T>(-2*dx_2Inv + k0_2*(pow(h_n[m], 2) - pow(nRef, 2)), 4*beta*dzInv);
    // checkCudaErrors(hipMemcpy(data.constLhsBot, &constLhsBot, T_size, hipMemcpyHostToDevice));
    // checkCudaErrors(hipMemcpy(data.constLhsTop, &constLhsTop, T_size, hipMemcpyHostToDevice));
    // checkCudaErrors(hipMemcpy(data.constRhsBot, &constRhsBot, T_size, hipMemcpyHostToDevice));
    // checkCudaErrors(hipMemcpy(data.constRhsTop, &constRhsTop, T_size, hipMemcpyHostToDevice));
    *(data.constLhsTop) = constLhsTop;
    *(data.constLhsBot) = constLhsBot;
    *(data.constRhsTop) = constRhsTop;
    *(data.constRhsBot) = constRhsBot;

    // setting first elements
    // first element in sub-diagonal is equal to 0 
    h_dl[0]   = cuGet<T>((T_REAL)0.0, (T_REAL)0.0); 
    h_d[0]    = cuFma(dx_2InvComplex, gammaLeft, constLhsTop);
    h_rhsUpdateArray[0] = constRhsTop;
    h_du[0]   = dx_2InvComplex;

    // setting last elements
    h_dl[m-1] = dx_2InvComplex;
    h_d[m-1]  = cuFma(dx_2InvComplex, gammaRight, constLhsBot);
    h_rhsUpdateArray[m-1] = constRhsBot;
    h_du[m-1] = cuGet<T>((T_REAL)0.0, (T_REAL)0.0);
    // last element in super diagonal is equal to 0
    
    // By following this convention, we can access elements of dl, du, d present in the same row by the row's index.

    h_b[0] = cuMul(cuFma(gammaLeft, dx_2InvComplex, constRhsTop), h_Ex[1]);
    h_b[0] = cuFma(dx_2InvComplex, h_Ex[2], h_b[0]);
    h_b[m-1] = cuMul(cuFma(gammaRight, dx_2InvComplex, constRhsTop), h_Ex[m-1]);
    h_b[m-1] = cuFma(dx_2InvComplex, h_Ex[m-2], h_b[m-1]);

    // setting interior elements
    for(int k=1; k<m-1; k++)
    {
        h_dl[k] = dx_2InvComplex;
        h_du[k] = dx_2InvComplex;
        h_d[k]  = cuGet<T>(2*dx_2Inv - k0_2*(pow(h_n[k+1], 2) - pow(nRef, 2)), 4*beta*dzInv);
        h_rhsUpdateArray[k]  = cuGet<T>(-2*dx_2Inv + k0_2*(pow(h_n[k+1], 2) - pow(nRef, 2)), 4*beta*dzInv);
        // h_b[k]  = cuGet<T>((-2*dx_2Inv + k0_2*(pow(h_n[k+1], 2) - pow(nRef, 2))) * cuReal(h_Ex[k+1]) - 4*beta*dzInv*cuImag(h_Ex[k+1]) + dx_2Inv * (cuReal(h_Ex[k]) + cuReal(h_Ex[k+2])), (-2*dx_2Inv + k0_2*(pow(h_n[k+1], 2) - pow(nRef, 2))) * cuImag(h_Ex[k+1]) + 4*beta*dzInv * cuReal(h_Ex[k+1]) + dx_2Inv * (cuImag(h_Ex[k]) + cuImag(h_Ex[k+2])));
        h_b[k]  = cuMul(cuGet<T>(dx_2InvComplex_1), h_Ex[k]);
        h_b[k]  = cuFma(cuGet<T>(-2*dx_2Inv + k0_2*(pow(h_n[k+1], 2) - pow(nRef, 2)), 4*beta*dzInv), h_Ex[k+1], h_b[k]);
        h_b[k]  = cuFma(cuGet<T>(dx_2InvComplex_1), h_Ex[k+2], h_b[k]);
    }
    
    // copying arrays from host to device
    checkCudaErrors(hipMemcpy(dl,  h_dl,   m*T_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d,   h_d,    m*T_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(du,  h_du,   m*T_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(b,   h_b,    m*T_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsUpdateArray,   h_rhsUpdateArray,    m*T_size, hipMemcpyHostToDevice));

    // setting device constant with value equal to dx_2InvComplex_1
    setConstants(&dx_2InvComplex_1);

    // finding 'marshaled' index of 1st, m-2 th, m-1 th element
    // 0 th elem remains in the same position
    int marshaledIndex_1;
    int marshaledIndex_m_2;
    int marshaledIndex_m_1;
    marshaledIndex_1   = find_marshaled_index<T, T_REAL>(&data, 1);
    marshaledIndex_m_2 = find_marshaled_index<T, T_REAL>(&data, m-2);
    marshaledIndex_m_1 = find_marshaled_index<T, T_REAL>(&data, m-1);
    data.setMarshaledIndex(marshaledIndex_1, marshaledIndex_m_2, marshaledIndex_m_1);

    // solving the matrix
    double start, stop;
    start = get_second();
    for(int i=0; i<steps; i++)
    {
        data.step = i;
        tridiagonalSolver<T, T_REAL>(&data, dl, d, du, b, bNew, rhsUpdateArray, m);
        hipDeviceSynchronize();
        hipGetLastError();
    }
    stop = get_second();
    printf("time on gpu = %.6f s\n", stop-start);

    // copy back the results to CPU
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(h_x_gpu, b, m*T_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_bNew_gpu, bNew, m*T_size, hipMemcpyDeviceToHost));
    
    start = get_second();
    for(int i=0; i<steps; i++)
    {
        data.step = i;
        tridiagonalSolverHost<T, T_REAL>(&data, h_dl, h_d, h_du, h_b, h_bNew, h_rhsUpdateArray, h_field, m);
    }
    stop = get_second();
    printf("time on cpu = %.6f s\n", stop-start);

    compare_result<T, T_REAL>(h_field, h_x_gpu, m, 1e-10, 1e-10, 50, stride);
    compare_result<T, T_REAL>(h_bNew, h_bNew_gpu, m, 1e-10, 1e-10, 50, stride);
    
    // Uncomment the next 12 lines only when program is being run without CPU computation
    // gammaLeft = cuDiv(h_x_gpu[0], h_x_gpu[1]);
    // gammaRight = cuDiv(h_x_gpu[m-1], h_x_gpu[m-2]);
    // h_rhsUpdateArray[0] = cuFma(gammaLeft, cuGet<T>(dx_2InvComplex_1), h_rhsUpdateArray[0]);    
    // h_rhsUpdateArray[m-1] = cuFma(gammaRight, cuGet<T>(dx_2InvComplex_1), h_rhsUpdateArray[m-1]);    

    // mv_test computes B (h_b_back) in Ax = B where x is the result from the gpu
    // mv_test<T>(h_b_back, h_dl, h_d, h_du, h_x_gpu, m);
    // mv_test_update<T>(h_bNew_back, cuGet<T>(dx_2InvComplex_1), h_rhsUpdateArray, h_x_gpu, m);

    // // compares the result from the gpu and the host
    // compare_result<T, T_REAL>(h_b, h_b_back, m, 1e-10, 1e-10, 50, stride);
    // compare_result<T, T_REAL>(h_bNew_gpu, h_bNew_back, m, 1e-10, 1e-10, 50, stride);

    for(int i=0; i < m; i++)
        fprintf(fp1, "%E\n", cuAbs(h_x_gpu[i]));
    

    checkCudaErrors(hipHostFree(h_d));
    checkCudaErrors(hipHostFree(h_b));
    checkCudaErrors(hipHostFree(h_n));
    checkCudaErrors(hipHostFree(h_x));
    checkCudaErrors(hipHostFree(h_dl));
    checkCudaErrors(hipHostFree(h_du));
    checkCudaErrors(hipHostFree(h_Ex));
    checkCudaErrors(hipHostFree(h_x_gpu));
    // checkCudaErrors(hipHostFree(h_b_back));
    checkCudaErrors(hipHostFree(h_bNew_gpu));
    // checkCudaErrors(hipHostFree(h_bNew_back));
    checkCudaErrors(hipHostFree(h_rhsUpdateArray));
    // TODO: don't forget to free memory
    // no need to find best grid every time, create buffers, just replace them and free them in this function.
    // use hipHostMalloc for everything --> pinned mem
    // check whether running kernel for dl, d, du is better or copying them again is better.
}

void
showHelp()
{
    printf("\nTridiagonal Solver : Command line options\n");
    printf("\t-device=n          (where n=0,1,2.... for the GPU device)\n\n");
    printf("> The default matrix size can be overridden with these parameters\n");
    printf("\t-size=row_dim_size (matrix row    dimensions)\n");
}

int 
main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help"))
    {
        showHelp();
        return 0;
    }

    // printf("\n-------------------------------------------\n");
    int m, steps, type, devID = findCudaDevice(argc, (const char **)argv);
    hipDeviceProp_t deviceProp;
    // Uncomment line 1011 in helper_cuda

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    // checkCudaErrors(hipSetDevice(0));
    // checkCudaErrors(hipGetDeviceProperties(&deviceProp, 0));

    // printf("> Device %d: \"%s\"\n", devID, deviceProp.name);
    // printf("> SM Capability %d.%d detected.\n", deviceProp.major, deviceProp.minor);
    
    if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        m = getCmdLineArgumentInt(argc, (const char **)argv, "size=");

        if (m < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(EXIT_FAILURE);
        }
        else
        {
            if (m < 10)
            {
                printf("Enter m value which is greater than 10. Exiting...\n");
                exit(EXIT_FAILURE);
            }
        }
    }
    else
        m = 1024;

    if (checkCmdLineFlag(argc, (const char **)argv, "steps"))
    {
        steps = getCmdLineArgumentInt(argc, (const char **)argv, "steps=");

        if (steps < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(EXIT_FAILURE);
        }
    }
    else
        steps = 1;

    if (checkCmdLineFlag(argc, (const char **)argv, "type"))
    {
        type = getCmdLineArgumentInt(argc, (const char **)argv, "type=");

        if (type < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(EXIT_FAILURE);
        }
    }
    else
        type = 1;

    // printf("-------------------------------------------\n");
    // printf("Matrix height = %d\n", m);
    // printf("-------------------------------------------\n");
    if(type == 1){
        // printf("GTSV solving using hipComplex ...\n");
        gtsv_randomMatrix<hipComplex, float>(m, steps);
        // printf("Finished GTSV solving using hipComplex\n");
    }
    if(type == 2){
        // printf("GTSV solving using hipDoubleComplex ...\n");
        gtsv_randomMatrix<hipDoubleComplex, double>(m, steps);
        // printf("Finished GTSV solving using hipDoubleComplex\n");
    }
    // printf("-------------------------------------------\n");

    return 0;
}
