#include "hip/hip_runtime.h"
/*******************************************************************************************************
                              University of Illinois/NCSA Open Source License
                                 Copyright (c) 2012 University of Illinois
                                          All rights reserved.

                                        Developed by: IMPACT Group
                                          University of Illinois
                                      http://impact.crhc.illinois.edu

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), 
to deal with the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense,
 and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

  Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimers.
  Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimers in the documentation and/or other materials provided with the distribution.
  Neither the names of IMPACT Group, University of Illinois, nor the names of its contributors may be used to endorse or promote products derived from this Software without specific prior written permission.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE
OR THE USE OR OTHER DEALINGS WITH THE SOFTWARE.

*******************************************************************************************************/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "cusparse_ops.hxx"
#include <stddef.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <sys/sysinfo.h>
#include <helper_string.h>    // helper for string parsing
#include <hip/hip_runtime_api.h>      // helper for cuda error checking functions

#define DEBUG 0

static double get_second (void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
}

template <typename T, typename T_REAL> 
void gtsv_spike_partial_diag_pivot(const T* dl, const T* d, const T* du, T* b,const int m,const int k);
template <typename T> 
void dtsvb_spike_v1(const T* dl, const T* d, const T* du, T* b,const int m);


//utility
#define EPS 1e-20

// mv_test fnxn takes tridiagonal matrix A (with diagonals a, b, c) and multiplies it with x (d) to give B (x)  
template <typename T> 
void mv_test
(
	T* x,				// result B
	const T* a,			// lower diagonal
	const T* b,			// diagonal
	const T* c,			// upper diagonal
	const T* d,			// variable vector
	const int sys_num,	// number of systems
	const int len,		// length of the matrix
	const int rhs 		// rhs
)
{
	printf("Multiplying A with result x to get B ...\n");
	int m=sys_num*len;
	
	for(int j=0;j<rhs;j++)
	{
		x[j*m] =  cuAdd( 	cuMul(b[0],d[j*m]), 
							cuMul(c[0],d[j*m+1]));
		// does the multiplication of the first row
		
		// multiplication of rows 1 to m-1
		for(int i=1; i<m-1; i++)
		{	
			//x[i]=  a[i]*d[i-1]+b[i]*d[i]+c[i]*d[i+1];
			x[j*m+i]=  cuMul(a[i], d[j*m+i-1]);
			x[j*m+i]=  cuFma(b[i], d[j*m+i], x[j*m+i]);
			// cuFma first multiplies 1st 2 params and then adds 3rd one  
			x[j*m+i]=  cuFma(c[i], d[j*m+i+1], x[j*m+i]);
		}
		
		// multiplication of last row m
		x[j*m+m-1]= cuAdd( cuMul(a[m-1],d[j*m+m-2]) , cuMul(b[m-1],d[j*m+m-1]) );
	}
	printf("Multiplication done.\n\n");
}


// compare_result<T, T_REAL>(h_b, h_b_back, 1, m, 1, 1e-10, 1e-10, 50, 3, b_dim);
template <typename T, typename T_REAL> 
void compare_result
(
	const T *x,				// B vector in Ax = B, given to us 
	const T *y,				// B vector in Ax = B, calc from GPU results 
	const int sys_num,		// number of systems
	const int len,			// length of matrix 
	const int rhs,			// num of rhs vectors
	const T_REAL abs_err,	// for abs error checking
	const T_REAL re_err,	// for rel error checking
	const int p_bound,		// bound on error counting
	const int k_bound,		// bound on RHS vector
	const int tx  			// 
)
{
	printf("Comparing computed B with given B.\n");
	int m = len*sys_num;
	
	T_REAL err = 0.0;
	T_REAL sum_err = 0.0;
	T_REAL total_sum = 0.0;
	T_REAL r_err = 1.0;
	T_REAL x_2 = 0.0;
	int p = 0; //error counter
	int t = 0; //check counter
	
	for(int k=0; k<rhs; k++)
	{
		if(k<k_bound)
		{
			printf("RHS vector is %d.\n",k);
			
		}
		p=0;
		for(int j=0;j<sys_num;j++)
		{
			if(k<k_bound)
				t=0;
			for(int i=0;i<len;i++)
			{
				T diff = cuSub(x[k*m+j*len+i], y[k*m+j*len+i]);
				err = cuReal(cuMul(diff, hipConj(diff) ));
				sum_err +=err;
				x_2 = cuReal(cuMul(x[k*m+j*len+i], hipConj(x[k*m+j*len+i])));
				total_sum += x_2;
				
				//avoid overflow in error check
				r_err = x_2 > EPS ? err/x_2:0.0;
				if(err > abs_err || r_err > re_err)
				{
					if(p < p_bound)
					{
						printf("Error occurred at system %d, element %2d, cpu = %10.6lf and gpu = %10.6lf at %d\n", j, i, cuReal(x[k*m+j*len+i]), cuReal(y[k*m+j*len+i]), i%tx);
						printf("Its absolute error is %le and relative error is %le.\n", err, r_err);
					}
					p++;
				}
				
				if(t < 16)
				{
					printf("Checked system %d, element %2d, cpu = %10.6lf and gpu = %10.6lf\n",j,i,cuReal(x[k*m+j*len+i]),cuReal(y[k*m+j*len+i]));
					t++;
				}
			}
		}
		if(k < k_bound)
		{
			if(p == 0)
			{
				printf("All correct.\n\n");
			}
			else
			{
				printf("There are %d errors.\n\n", p);
			}
		}
	}
	printf("Total absolute error is %le\n",sqrt(sum_err));
	printf("Total relative error is %le\n",sqrt(sum_err)/sqrt(total_sum));
	printf("Comparing done.\n\n");
}

//This is a testing gtsv function
template <typename T, typename T_REAL> 
void test_gtsv_v1(int m)
{
	double start, stop; // timers
	
	// each array is a set of elements in a diagonal stored in contiguous mem locations.
	T *h_dl; 	//	set of lower diagonal elements of mat A (n-1 elements)
	T *h_d; 	//	diagonal elements of mat A (n elements)
	T *h_du; 	//	set of upper diagonal elements of mat A (n-1 elements)
	T *h_b;		// 	RHS array has n elements
	
	T *h_x_gpu;	//	results from GPU
	T *h_b_back;// 

	// vectors on the device
	T *dl; 
	T *d;
	T *du;
	T *b;

	// allocation
	// the vectors on the device are all set to zero
	{
		h_dl=(T *)malloc(sizeof(T)*m);
		h_du=(T *)malloc(sizeof(T)*m);
		h_d=(T *)malloc(sizeof(T)*m);
		h_b=(T *)malloc(sizeof(T)*m);
		
		h_x_gpu=(T *)malloc(sizeof(T)*m);
		h_b_back=(T *)malloc(sizeof(T)*m);
				
		hipMalloc((void **)&dl, sizeof(T)*m); 
		hipMalloc((void **)&du, sizeof(T)*m); 
		hipMalloc((void **)&d, sizeof(T)*m); 
		hipMalloc((void **)&b, sizeof(T)*m); 

		hipMemset(d, 0, m * sizeof(T));
		hipMemset(dl, 0, m * sizeof(T));
		hipMemset(du, 0, m * sizeof(T));
	}
	
	srand(54321);

	// used for random number generation
	// max value returned by srand is stored in RAND_MAX 
	// generate random data
	h_dl[0]   = cuGet<T>(0); 
	// first elemenyt in sub-diagonal is equal to 0 
	h_d[0]    = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0 );
	h_du[0]   = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	h_dl[m-1] = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	h_d[m-1]  = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	h_du[m-1] = cuGet<T>(0); 
	// last element in super diagonal is equal to 0
	// By following this convention, we can access elements of dl, du, d present in the same row by the row's index.

	h_b[0]    = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0 );
	h_b[m-1]  = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0 );
	
	for(int k = 1; k < m-1; k++)
	{
		h_dl[k] =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
		h_du[k] =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
		h_d[k]  =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
		h_b[k]  =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	}
	
	
	// Memory copy from host to device
	hipMemcpy(dl, h_dl, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d, h_d, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(du, h_du, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(b, h_b, m*sizeof(T), hipMemcpyHostToDevice);

	// solving a general matrix
	// noting the time stamps
    start = get_second();
    gtsv_spike_partial_diag_pivot<T,T_REAL>( dl, d, du, b, m, 1);
    // last parameter is used to run v1
    hipDeviceSynchronize();
	stop = get_second();

    printf("time = %.6f s\n\n", stop-start);    

  	//copy back the results to CPU
	hipMemcpy(h_x_gpu, b, m*sizeof(T), hipMemcpyDeviceToHost);

    // mv_test computes B (h_b_back) in Ax = B where x is the reult from the gpu
    mv_test<T>(h_b_back, h_dl, h_d, h_du, h_x_gpu, 1, m, 1);
    
    // backward error check
	int b_dim = 128;  
	// this value is type specific. here double for setting it's value, look at bestGridDim in spike_host.cu

	// compares the result from the gpu and the host
	compare_result<T, T_REAL>(h_b, h_b_back, 1, m, 1, 1e-10, 1e-10, 50, 3, b_dim);
}




//This is a testing gtsv function
template <typename T, typename T_REAL> 
void test_gtsv_v_few(int m,int rhs)
{
	double start,stop;
	T *h_dl;
	T *h_d;
	T *h_du;
	T *h_b;
	
	T *h_x_gpu;
	T *h_b_back;

	T *dl;
	T *d;
	T *du;
	T *b;

	
	//allocation
	{
		h_dl=(T *)malloc(sizeof(T)*m);
		h_du=(T *)malloc(sizeof(T)*m);
		h_d=(T *)malloc(sizeof(T)*m);
		h_b=(T *)malloc(sizeof(T)*m*rhs);
		
		h_x_gpu=(T *)malloc(sizeof(T)*m*rhs);
		h_b_back=(T *)malloc(sizeof(T)*m*rhs);
				
		hipMalloc((void **)&dl, sizeof(T)*m); 
		hipMalloc((void **)&du, sizeof(T)*m); 
		hipMalloc((void **)&d, sizeof(T)*m); 
		hipMalloc((void **)&b, sizeof(T)*m*rhs); 

		hipMemset(d, 0, m * sizeof(T));
		hipMemset(dl, 0, m * sizeof(T));
		hipMemset(du, 0, m * sizeof(T));
	}
	

#if DEBUG
	int k;
	//generate random data
	h_dl[0]   = cuGet<T>(0);   //always 0
	h_d[0]    = cuGet<T>(2);
	h_du[0]   = cuGet<T>(1);
	h_dl[m-1] = cuGet<T>(0);
	h_d[m-1]  = cuGet<T>(2);
	h_du[m-1] = cuGet<T>(0); //always 0
	for(k=1;k<m-1;k++)
	{
		h_dl[k] =cuGet<T>(0);
		h_d[k]  =cuGet<T>(2);
		h_du[k] =cuGet<T>(1);
		
	}
	
	
	for(int i=0;i<rhs;i++)
		for(k=0;k<m;k++)
			h_b[i*m+k]  =cuGet<T>(k);

#else
	int k;
	srand(54321);
	//generate random data
	h_dl[0]   = cuGet<T>(0);//always 0
	h_d[0]    = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0 );
	h_du[0]   = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	h_dl[m-1] = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	h_d[m-1]  = cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	h_du[m-1] = cuGet<T>(0);//always 0
	for(k=1;k<m-1;k++)
	{
		h_dl[k] =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
		h_d[k]  =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
		h_du[k] =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);
	}
	
	
	for(int i=0;i<rhs;i++)
		for(k=0;k<m;k++)
			h_b[i*m+k]  =cuGet<T>( (rand()/(double)RAND_MAX)*2.0-1.0);

#endif
	
	
   //Memory copy
	hipMemcpy(dl, h_dl, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d, h_d, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(du, h_du, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(b, h_b, m*sizeof(T)*rhs, hipMemcpyHostToDevice);

	//this is for general matrix
    start = get_second();
    // goes to spike_host:357
    gtsv_spike_partial_diag_pivot<T,T_REAL>( dl, d, du, b, m, rhs);
    
    hipDeviceSynchronize();
	stop = get_second();
    printf("time = %.6f s\n\n", stop-start);    
	
  	//copy back 
	hipMemcpy(h_x_gpu, b, m*sizeof(T)*rhs, hipMemcpyDeviceToHost);

    mv_test<T>(h_b_back,h_dl,h_d,h_du,h_x_gpu,1,m,rhs);
    //backward error check
	int b_dim=128;  
	//for debug
	compare_result<T,T_REAL>(h_b,h_b_back,1,m,rhs,1e-10,1e-10,50,3,b_dim);
}




//This is a testing gtsv function
template <typename T, typename T_REAL> 
void test_dtsvb_v1(int m)
{
	double start,stop;
	T *h_dl;
	T *h_d;
	T *h_du;
	T *h_b;
	
	T *h_x_gpu;
	T *h_b_back;

	T *dl;
	T *d;
	T *du;
	T *b;

	
	//allocation
	{
		h_dl=(T *)malloc(sizeof(T)*m);
		h_du=(T *)malloc(sizeof(T)*m);
		h_d=(T *)malloc(sizeof(T)*m);
		h_b=(T *)malloc(sizeof(T)*m);
		
		h_x_gpu=(T *)malloc(sizeof(T)*m);
		h_b_back=(T *)malloc(sizeof(T)*m);
				
		hipMalloc((void **)&dl, sizeof(T)*m); 
		hipMalloc((void **)&du, sizeof(T)*m); 
		hipMalloc((void **)&d, sizeof(T)*m); 
		hipMalloc((void **)&b, sizeof(T)*m); 

		hipMemset(d, 0, m * sizeof(T));
		hipMemset(dl, 0, m * sizeof(T));
		hipMemset(du, 0, m * sizeof(T));
	}
	

	
	int k;
	srand(54321);
	//generate random data
	h_dl[0]= cuGet<T>(0);
	h_du[0]= cuGet<T>( (rand()/(double)RAND_MAX) );
	h_d[0]= cuMul(cuAdd(h_dl[0],h_du[0]),cuGet<T>(2));
	h_dl[m-1]=cuGet<T>( (rand()/(double)RAND_MAX) );
	h_du[m-1]=cuGet<T>(0);
	h_d[m-1]= cuMul(cuAdd(h_dl[m-1],h_du[m-1]),cuGet<T>(2));
	h_b[0]=cuGet<T>( (rand()/(double)RAND_MAX) );
	h_b[m-1]=cuGet<T>( (rand()/(double)RAND_MAX) );
	
	for(k=1;k<m-1;k++)
	{
		h_dl[k]=cuGet<T>( (rand()/(double)RAND_MAX) );
		h_du[k]=cuGet<T>( (rand()/(double)RAND_MAX) );
		h_d[k]= cuMul(cuAdd(h_dl[k],h_du[k]),cuGet<T>(2));
		h_b[k]=cuGet<T>( (rand()/(double)RAND_MAX) );
	}
	
	
   //Memory copy
	hipMemcpy(dl, h_dl, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d, h_d, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(du, h_du, m*sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(b, h_b, m*sizeof(T), hipMemcpyHostToDevice);

	//this is for general matrix
    start = get_second();
    dtsvb_spike_v1<T>( dl, d, du, b,m);
    hipDeviceSynchronize();
	stop = get_second();
    printf("test_gtsv_v1 m=%d time=%.6f\n", m, stop-start);
    


  	//copy back 
	hipMemcpy(h_x_gpu, b, m*sizeof(T), hipMemcpyDeviceToHost);

    mv_test<T>(h_b_back,h_dl,h_d,h_du,h_x_gpu,1,m,1);
    //backward error check
	int b_dim=128;  //for debug
	compare_result<T,T_REAL>(h_b,h_b_back,1,m,1,1e-10,1e-10,50,3,b_dim);
}

void
showHelp()
{
    printf("\nTridiagonal Solver : Command line options\n");
    printf("\t-device=n          (where n=0,1,2.... for the GPU device)\n\n");
    printf("> The default matrix size can be overridden with these parameters\n");
    printf("\t-size=row_dim_size (matrix row    dimensions)\n");
    printf("\t-rhs=number_of_rhs_vectors\n");
}

int main(int argc, char **argv)
{
	if (checkCmdLineFlag(argc, (const char **)argv, "help"))
    {
        showHelp();
        return 0;
    }

    int m, k, devID = findCudaDevice(argc, (const char **)argv);
    hipDeviceProp_t deviceProp;

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
	
	printf("> Device %d: \"%s\"\n", devID, deviceProp.name);
    printf("> SM Capability %d.%d detected.\n", deviceProp.major, deviceProp.minor);
	
    if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        m = getCmdLineArgumentInt(argc, (const char **)argv, "size=");

        if (m < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(EXIT_FAILURE);
        }
        else
        {
            if (m < 10)
            {
                printf("Enter m value which is greater than 10. Exiting...\n");
                exit(EXIT_FAILURE);
            }
        }
    }
    else
        m = 512*1024+512;

    if (checkCmdLineFlag(argc, (const char **)argv, "rhs"))
    {
        k = getCmdLineArgumentInt(argc, (const char **)argv, "rhs=");

        if (k < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(EXIT_FAILURE);
        }
        else
        {
            if (k > 3)
            {
                printf("k value should be less than 3. Exiting...\n");
                exit(EXIT_FAILURE);
            }
        }
    }
    else
        k = 2;

	printf("\nmatrix size = %d and rhs is %d \n", m, k);
    
	printf("-------------------------------------------\n");
	printf("GTSV testing using double ...\n");
	test_gtsv_v1<double, double>(m);	
    printf("Finished GTSV testing using double\n\n");
	printf("-------------------------------------------\n");
	exit(1);
	printf("GTSV testing using double and multiple RHS ...\n");
	test_gtsv_v_few<double,double>(m,k);
    printf("Finished GTSV testing multiple RHS\n\n");
	
	/*
    printf("Double complex test_gtsv 5 rhs testing\n");    
	test_gtsv_v_few<hipDoubleComplex,double>(m,5);    
    printf("END Double complex test_gtsv 5 rhs\n");
    
	
	
	printf("double test_dtsvb_v1 testing\n");
	test_dtsvb_v1<double,double>(m);
	
	printf("double complex test_dtsvb_v1 testing\n");
	test_dtsvb_v1<hipDoubleComplex,double>(m);
	
	*/	
	//printf("float testing\n");
	//test_dtsvb_v1<float>(m);
  
	return 0;

}
