/*******************************************************************************************************
                              University of Illinois/NCSA Open Source License
                                 Copyright (c) 2012 University of Illinois
                                          All rights reserved.

                                        Developed by: IMPACT Group
                                          University of Illinois
                                      http://impact.crhc.illinois.edu

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal with the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

  Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimers.
  Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimers in the documentation and/or other materials provided with the distribution.
  Neither the names of IMPACT Group, University of Illinois, nor the names of its contributors may be used to endorse or promote products derived from this Software without specific prior written permission.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH THE SOFTWARE.

*******************************************************************************************************/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include "datablock.h"
#include "spike_kernel.hxx"
#include "cusparse_ops.hxx"
#include <complex.h>

//template <typename T>
void setConstants(hipDoubleComplex *dx_2InvNeg)
{
     checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(constant1), dx_2InvNeg,
                    sizeof(hipDoubleComplex)));
     checkCudaErrors(hipGetLastError());
}

template <typename T, typename T_REAL> 
void tridiagonalSolver(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T* bNew, T* rhsUpdateArray, const int m)
{
    // prefer larger L1 cache and smaller shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tiled_diag_pivot_x1<T),T_REAL>, hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(spike_GPU_back_sub_x1<T), T_REAL>, hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(multiply_kernel<T>), hipFuncCachePreferL1);
    
    T* dl_buffer    = data->dl_buffer;    // lower digonal after DM
    T* d_buffer     = data->d_buffer;     // diagonal after DM
    T* du_buffer    = data->du_buffer;    // upper diagonal after DM
    T* b_buffer     = data->b_buffer;     // B array after DM (here, B is in Ax = B)
    T* w_buffer     = data->w_buffer;     // W in A_i * W_i = vector w/ partition's lower diagonal element
    T* v_buffer     = data->v_buffer;     // V in A_i * V_i = vector w/ partition's upper diagonal element
    T* c2_buffer    = data->c2_buffer;    // stores modified diagonal elements in diagonal pivoting method
    T* bNew_buffer  = data->bNew_buffer;  // new DM B array after multiplying with updated A (here, B is in Ax = B) 
    T* rhsUpdateArrayBuffer  = data->rhsUpdateArrayBuffer;  // DM RHS update array
    T* bottomElemBuffer      = data->bottomElemBuffer;      // elements needed for finding new rhs' bottom elems
    T* topElemBuffer         = data->topElemBuffer;         // elements needed for finding new rhs' top elems
    
    T* x_level_2 = data->x_level_2;
    T* w_level_2 = data->w_level_2;
    T* v_level_2 = data->v_level_2;
    
    int step = data->step;
    T_REAL* field = data->field;
    size_t pitch = data->pitch;

    int local_reduction_share_size  = data->local_reduction_share_size;
    int global_share_size           = data->global_share_size;
    int local_solving_share_size    = data->local_solving_share_size;
    int marshaling_share_size       = data->marshaling_share_size;

    dim3 gridDim  = data->gridDim;
    dim3 blockDim = data->blockDim;

    int s       = data->s;
    int b_dim   = data->b_dim;
    int stride  = data->h_stride;
    int tile    = 128;

    int marshaledIndex_1    = data->marshaledIndex_1;
    int marshaledIndex_m_2  = data->marshaledIndex_m_2;
    int marshaledIndex_m_1  = data->marshaledIndex_m_1;

    T_REAL dx = *(data->dx);
    T *h_x_0    = data->h_x_0;
    T *h_x_1    = data->h_x_1;
    T *h_x_m_2  = data->h_x_m_2;
    T *h_x_m_1  = data->h_x_m_1;
    T *h_diagonal_m_1   = data->h_diagonal_m_1;
    T *h_diagonal_0     = data->h_diagonal_0;

    T* h_gammaLeft      = data->h_gammaLeft;
    T* h_kxbLeft        = data->h_kxbLeft;
    T* h_gammaRight     = data->h_gammaRight;
    T* h_kxbRight       = data->h_kxbRight;
    T* dx_2InvNeg       = data->dx_2InvNeg;   // equals -1/(dx*dx)
    T* dx_2InvPos       = data->dx_2InvPos;   // equals +1/(dx*dx)
    checkCudaErrors(hipMemset(bNew_buffer, 0, sizeof(T)*s*b_dim*stride));

    // data layout transformation
    if(data->step == 0){
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(dl_buffer, dl, stride, b_dim, m, cuGet<T>(0));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(du_buffer, du, stride, b_dim, m, cuGet<T>(0));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(rhsUpdateArrayBuffer,  rhsUpdateArray,  stride, b_dim, m, cuGet<T>(1));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(d_buffer,  d,  stride, b_dim, m, cuGet<T>(1));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(b_buffer,  b,  stride, b_dim, m, cuGet<T>(0));
    }

    // partitioned solver
    tiled_diag_pivot_x1<T,T_REAL><<<s, b_dim>>>(b_buffer, w_buffer, v_buffer, c2_buffer, dl_buffer, d_buffer, du_buffer, stride, tile);
    
    // SPIKE solver
    spike_local_reduction_x1<T><<<s, b_dim, local_reduction_share_size>>>(b_buffer, w_buffer, v_buffer, x_level_2, w_level_2, v_level_2, stride);
    spike_GPU_global_solving_x1<<<1, 32, global_share_size>>>(x_level_2, w_level_2, v_level_2, s);
    spike_GPU_local_solving_x1<T><<<s, b_dim, local_solving_share_size>>>(b_buffer, w_buffer, v_buffer, x_level_2, stride);
    spike_GPU_back_sub_x1<T, T_REAL><<<s, b_dim>>>(b_buffer, w_buffer, v_buffer, x_level_2, stride, field + step*pitch/sizeof(T_REAL));
    // Solution to Ax = B is in b_buffer. It is data marshaled here.
    
    checkCudaErrors(hipMemcpy(h_x_0,   b_buffer,                    sizeof(T), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_x_1,   b_buffer+marshaledIndex_1,   sizeof(T), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_x_m_2, b_buffer+marshaledIndex_m_2, sizeof(T), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_x_m_1, b_buffer+marshaledIndex_m_1, sizeof(T), hipMemcpyDeviceToHost));

    *h_gammaLeft     = cuDiv(*h_x_0, *h_x_1);
    *h_gammaRight    = cuDiv(*h_x_m_1, *h_x_m_2);
    
    *h_kxbLeft = cuDiv(cuMul(cuLog(*h_gammaLeft), cuGet<T>((T_REAL)0.0, (T_REAL)1.0)), cuGet<T>(dx, (T_REAL)0.0));
    if(cuReal(*h_kxbLeft) < 0){
        *h_kxbLeft = cuGet<T>((T_REAL)0.0, cuImag(*h_kxbLeft));
        *h_gammaLeft = cuExp(cuMul(cuGet<T>((T_REAL)0.0, -dx), *h_kxbLeft));
    }
    
    *h_kxbRight = cuDiv(cuMul(cuLog(*h_gammaRight), cuGet<T>((T_REAL)0.0, (T_REAL)1.0)), cuGet<T>(dx, (T_REAL)0.0));
    if(cuReal(*h_kxbRight) < 0){
        *h_kxbRight = cuGet<T>((T_REAL)0.0, cuImag(*h_kxbRight));
        *h_gammaRight = cuExp(cuMul(cuGet<T>((T_REAL)0.0, -dx), *h_kxbRight));
    }

    *h_diagonal_0    = cuFma(*h_gammaLeft, *dx_2InvPos, *(data->constRhsTop));
    *h_diagonal_m_1  = cuFma(*h_gammaRight, *dx_2InvPos, *(data->constRhsBot));

    checkCudaErrors(hipMemcpy(rhsUpdateArrayBuffer, h_diagonal_0, sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsUpdateArrayBuffer+marshaledIndex_m_1, h_diagonal_m_1, sizeof(T), hipMemcpyHostToDevice));

    // TODO: time this thing on GPU/CPU? Check this.
    int blockSize = b_dim*stride;
    b_buffer += b_dim*(stride-1);
    topElemBuffer += 1;
    for (int i=0; i<s; i++)
        checkCudaErrors(hipMemcpy(topElemBuffer + i*b_dim, b_buffer + i*blockSize, sizeof(T)*b_dim, hipMemcpyDeviceToDevice));

    b_buffer -= b_dim*(stride-1);
    topElemBuffer -= 1;
    checkCudaErrors(hipMemset(topElemBuffer, 0, sizeof(T)));

    for (int i=0; i<s; i++)
        checkCudaErrors(hipMemcpy(bottomElemBuffer + i*b_dim, b_buffer + i*blockSize, sizeof(T)*b_dim, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipMemset(bottomElemBuffer + s*(b_dim), 0, sizeof(T)));

    // finds new RHS with rhsUpdateArrayBuffer having its 1st and last elements modified
    multiply_kernel<T><<<s, b_dim>>>(rhsUpdateArrayBuffer, topElemBuffer, bottomElemBuffer+1, b_buffer, bNew_buffer, stride, tile);

    // do back data marshaling only in the last step
    if(step == (data->totalSteps)-1){
        back_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(b, b_buffer, stride, b_dim, m);
        back_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(bNew, bNew_buffer, stride, b_dim, m);
        // may not be reqd.! check!!!
    }
    
    // updating A in Ax=B. This will be used again in the next step for solving. 
    else{
        checkCudaErrors(hipMemcpy(b_buffer, bNew_buffer, sizeof(T)*data->mPad, hipMemcpyDeviceToDevice));
        
        // modifying main diagonal
        *h_diagonal_0    = cuFma(*h_gammaLeft, *dx_2InvNeg, *(data->constLhsTop));
        *h_diagonal_m_1  = cuFma(*h_gammaRight, *dx_2InvNeg, *(data->constLhsBot));
        
        checkCudaErrors(hipMemcpy(d_buffer, h_diagonal_0, sizeof(T), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_buffer+marshaledIndex_m_1, h_diagonal_m_1, sizeof(T), hipMemcpyHostToDevice));
    }
    
    // hipMemcpy(h_gammaLeft, d_gamma)
    // printf("Solving done.\n\n");
    // free pivotingData both h and dev
    // use checkCudaErrors for all cudaMallocs
    // change all *h_x to h_x
}

template <typename T, typename T_REAL> 
void tridiagonalSolverHost(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T* bNew, T* rhsUpdateArray, T* x, const int m)
{
    T* h_gammaLeft  = data->h_gammaLeft;
    T* h_gammaRight = data->h_gammaRight;
    T* h_kxbLeft    = data->h_kxbLeft;
    T* h_kxbRight   = data->h_kxbRight;
    T* dx_2InvNeg   = data->dx_2InvNeg;
    T* dx_2InvPos   = data->dx_2InvPos;
    T_REAL dx = *(data->dx);
    T *gamma  = data->gamma;
    T *h_diagonal_m_1   = data->h_diagonal_m_1;
    T *h_diagonal_0     = data->h_diagonal_0;
    T  beta = d[0];
    x[0] = cuDiv(b[0], beta);
    int i;
    for (i=1; i<m; i++){
        gamma[i] = cuDiv(du[i-1], beta);
        beta = cuFma(cuNeg(gamma[i]), dl[i], d[i]);
        x[i] = cuFma(cuNeg(x[i-1]), dl[i], b[i]);
        x[i] = cuDiv(x[i], beta);
    }
    int k;
    for (i=1; i<m; i++){
        k = m-i;
        x[k-1] = cuFma(cuNeg(x[k]), gamma[k], x[k-1]);
    }

    *h_gammaLeft     = cuDiv(x[0], x[1]);
    *h_gammaRight    = cuDiv(x[m-1], x[m-2]);
    
    *h_kxbLeft = cuDiv(cuMul(cuLog(*h_gammaLeft), cuGet<T>((T_REAL)0.0, (T_REAL)1.0)), cuGet<T>(dx, (T_REAL)0.0));
    if(cuReal(*h_kxbLeft) < 0){
        *h_kxbLeft = cuGet<T>((T_REAL)0.0, cuImag(*h_kxbLeft));
        *h_gammaLeft = cuExp(cuMul(cuGet<T>((T_REAL)0.0, -dx), *h_kxbLeft));
    }
    
    *h_kxbRight = cuDiv(cuMul(cuLog(*h_gammaRight), cuGet<T>((T_REAL)0.0, (T_REAL)1.0)), cuGet<T>(dx, (T_REAL)0.0));
    if(cuReal(*h_kxbRight) < 0){
        *h_kxbRight = cuGet<T>((T_REAL)0.0, cuImag(*h_kxbRight));
        *h_gammaRight = cuExp(cuMul(cuGet<T>((T_REAL)0.0, -dx), *h_kxbRight));
    }
    
    *h_diagonal_0    = cuFma(*h_gammaLeft, *dx_2InvPos, *(data->constRhsTop));
    *h_diagonal_m_1  = cuFma(*h_gammaRight, *dx_2InvPos, *(data->constRhsBot));
    rhsUpdateArray[0] = *h_diagonal_0;
    rhsUpdateArray[m-1] = *h_diagonal_m_1;
    
    bNew[0]   = cuAdd(cuMul(rhsUpdateArray[0], x[0]), cuMul(*dx_2InvPos, x[1]));
    bNew[m-1] = cuAdd(cuMul(*dx_2InvPos, x[m-2]), cuMul(rhsUpdateArray[m-1], x[m-1]));
    for (i=1; i<m-1; i++){
        bNew[i] = cuMul(*dx_2InvPos, x[i-1]);
        bNew[i] = cuFma(rhsUpdateArray[i], x[i], bNew[i]);
        bNew[i] = cuFma(*dx_2InvPos, x[i+1], bNew[i]);
    }

    if (data->step != (data->totalSteps-1)){
        d[0] = cuFma(*h_gammaLeft, *dx_2InvNeg, *(data->constLhsTop));
        d[m-1] = cuFma(*h_gammaRight, *dx_2InvNeg, *(data->constLhsBot));
        memcpy(b, bNew, sizeof(T)*m);
    }
}

// template<typename T>
// void set_constants(T *dx_2InvNeg);
// template void set_constants<hipComplex>(hipComplex *);
// template 
// void set_constants<hipDoubleComplex>(hipDoubleComplex *);

template <typename T, typename T_REAL> 
void tridiagonalSolver(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T *bNew, T *rhsUpdateArray, const int m);
/* explicit instanciation */
template void tridiagonalSolver<hipComplex, float>(Datablock<hipComplex, float> *data, const hipComplex* dl, hipComplex* d, const hipComplex* du, hipComplex* b, hipComplex *bNew, hipComplex *rhsUpdateArray, const int m);
template void tridiagonalSolver<hipDoubleComplex, double>(Datablock<hipDoubleComplex, double> *data, const hipDoubleComplex* dl, hipDoubleComplex* d, const hipDoubleComplex* du, hipDoubleComplex* b, hipDoubleComplex *bNew, hipDoubleComplex *rhsUpdateArray, const int m);


template <typename T, typename T_REAL> 
void tridiagonalSolverHost(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T *bNew, T *rhsUpdateArray, T* x, const int m);
// explicit instanciation
template void tridiagonalSolverHost<hipComplex, float>(Datablock<hipComplex, float> *data, const hipComplex* dl, hipComplex* d, const hipComplex* du, hipComplex* b, hipComplex *bNew, hipComplex *rhsUpdateArray, hipComplex *x, const int m);
template void tridiagonalSolverHost<hipDoubleComplex, double>(Datablock<hipDoubleComplex, double> *data, const hipDoubleComplex* dl, hipDoubleComplex* d, const hipDoubleComplex* du, hipDoubleComplex* b, hipDoubleComplex *bNew, hipDoubleComplex *rhsUpdateArray, hipDoubleComplex *x, const int m);