/*******************************************************************************************************
                              University of Illinois/NCSA Open Source License
                                 Copyright (c) 2012 University of Illinois
                                          All rights reserved.

                                        Developed by: IMPACT Group
                                          University of Illinois
                                      http://impact.crhc.illinois.edu

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal with the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

  Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimers.
  Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimers in the documentation and/or other materials provided with the distribution.
  Neither the names of IMPACT Group, University of Illinois, nor the names of its contributors may be used to endorse or promote products derived from this Software without specific prior written permission.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH THE SOFTWARE.

*******************************************************************************************************/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include "datablock.h"
#include "spike_kernel.hxx"
#include "cusparse_ops.hxx"

//template <typename T>
void setConstants(hipDoubleComplex *dx_2InvComplex)
{
     checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(constant1), dx_2InvComplex,
                    sizeof(hipDoubleComplex)));
     checkCudaErrors(hipGetLastError());
}

template <typename T, typename T_REAL> 
void tridiagonalSolver(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T* bNew, T* rhsUpdateArray, const int m)
{
    // prefer larger L1 cache and smaller shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tiled_diag_pivot_x1<T),T_REAL>,hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(spike_GPU_back_sub_x1<T>),hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(multiply_kernel<T>),hipFuncCachePreferL1);
    
    // variables for finding no of 1 by 1 pivotings
    // int *h_pivotingData;
    // int *pivotingData;

    // h_pivotingData = (int *)malloc(sizeof(int));
    // checkCudaErrors(hipMalloc((void **)&pivotingData, sizeof(int)));
    // checkCudaErrors(hipMemset((void *)pivotingData, 0, sizeof(int)));

    T* dl_buffer    = data->dl_buffer;    // lower digonal after DM
    T* d_buffer     = data->d_buffer;     // diagonal after DM
    T* du_buffer    = data->du_buffer;    // upper diagonal after DM
    T* b_buffer     = data->b_buffer;     // B array after DM (here, B is in Ax = B)
    T* w_buffer     = data->w_buffer;     // W in A_i * W_i = vector w/ partition's lower diagonal element
    T* v_buffer     = data->v_buffer;     // V in A_i * V_i = vector w/ partition's upper diagonal element
    T* c2_buffer    = data->c2_buffer;    // stores modified diagonal elements in diagonal pivoting method
    T* bNew_buffer  = data->bNew_buffer;  // new DM B array after multiplying with updated A (here, B is in Ax = B) 
    T* rhsUpdateArrayBuffer  = data->rhsUpdateArrayBuffer;  // DM RHS update array
    T* bottomElemBuffer      = data->bottomElemBuffer;
    T* topElemBuffer         = data->topElemBuffer;
    bool *flag               = data->flag;
    
    T* x_level_2 = data->x_level_2;
    T* w_level_2 = data->w_level_2;
    T* v_level_2 = data->v_level_2;

    int local_reduction_share_size  = data->local_reduction_share_size;     
    int global_share_size           = data->global_share_size;      
    int local_solving_share_size    = data->local_solving_share_size;
    int marshaling_share_size       = data->marshaling_share_size;

    dim3 gridDim  = data->gridDim;
    dim3 blockDim = data->blockDim;

    int s       = data->s;
    int b_dim   = data->b_dim;
    int stride  = data->h_stride;
    int tile    = 128;

    // int marshaledIndex_1    = data->marshaledIndex_1;
    // int marshaledIndex_m_2  = data->marshaledIndex_m_2;
    int marshaledIndex_m_1  = data->marshaledIndex_m_1;
    
    // T *h_x_0    = data->h_x_0;
    // T *h_x_1    = data->h_x_1;
    // T *h_x_m_2  = data->h_x_m_2;
    // T *h_x_m_1  = data->h_x_m_1;
    T *h_diagonal_m_1  = data->h_diagonal_m_1;
    T *h_diagonal_0  = data->h_diagonal_0;

    T* h_gammaLeft      = data->h_gammaLeft;
    T* h_gammaRight     = data->h_gammaRight;
    T* dx_2InvComplex   = data->dx_2InvComplex;     // equals -1/(dx*dx)
    T* dx_2InvComplex_1 = data->dx_2InvComplex_1;   // equals +1/(dx*dx)

    // kernels 
    // data layout transformation
    if(data->step == 0){
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(dl_buffer, dl, stride, b_dim, m, cuGet<T>(0));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(du_buffer, du, stride, b_dim, m, cuGet<T>(0));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(rhsUpdateArrayBuffer,  rhsUpdateArray,  stride, b_dim, m, cuGet<T>(0));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(d_buffer,  d,  stride, b_dim, m, cuGet<T>(1));
        forward_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(b_buffer,  b,  stride, b_dim, m, cuGet<T>(0));
    }

    // partitioned solver
    tiled_diag_pivot_x1<T,T_REAL><<<s, b_dim>>>(b_buffer, w_buffer, v_buffer, c2_buffer, flag, dl_buffer, d_buffer, du_buffer, stride, tile);
    
    // SPIKE solver
    spike_local_reduction_x1<T><<<s, b_dim, local_reduction_share_size>>>(b_buffer, w_buffer, v_buffer, x_level_2, w_level_2, v_level_2, stride);
    spike_GPU_global_solving_x1<<<1, 32, global_share_size>>>(x_level_2, w_level_2, v_level_2, s);
    spike_GPU_local_solving_x1<T><<<s, b_dim, local_solving_share_size>>>(b_buffer, w_buffer, v_buffer, x_level_2, stride);
    spike_GPU_back_sub_x1<T><<<s, b_dim>>>(b_buffer, w_buffer, v_buffer, x_level_2, stride);
    // Solution to Ax = B is in b_buffer. It is data marshaled here.

    // checkCudaErrors(hipMemcpy(h_x_0,   b_buffer,                    sizeof(T), hipMemcpyDeviceToHost));
    // checkCudaErrors(hipMemcpy(h_x_1,   b_buffer+marshaledIndex_1,   sizeof(T), hipMemcpyDeviceToHost));
    // checkCudaErrors(hipMemcpy(h_x_m_2, b_buffer+marshaledIndex_m_2, sizeof(T), hipMemcpyDeviceToHost));
    // checkCudaErrors(hipMemcpy(h_x_m_1, b_buffer+marshaledIndex_m_1, sizeof(T), hipMemcpyDeviceToHost));
    // printf("h_x_0   = %E.\n", cuAbs(*h_x_0));
    // printf("h_x_1   = %E.\n", cuAbs(*h_x_1));
    // printf("h_x_m_2 = %E.\n", cuAbs(*h_x_m_2));
    // printf("h_x_m_1 = %E.\n", cuAbs(*h_x_m_1));
    // *h_gammaLeft     = cuDiv(*h_x_0, *h_x_1);
    // *h_gammaRight    = cuDiv(*h_x_m_1, *h_x_m_2);
    *h_gammaLeft     = cuGet<T>((T_REAL)0.0, (T_REAL)0.0);
    *h_gammaRight    = cuGet<T>((T_REAL)0.0, (T_REAL)0.0);
    *h_diagonal_0    = cuFma(*h_gammaLeft, *dx_2InvComplex_1, *(data->constRhsTop));
    *h_diagonal_m_1  = cuFma(*h_gammaRight, *dx_2InvComplex_1, *(data->constRhsBot));

    checkCudaErrors(hipMemcpy(rhsUpdateArrayBuffer, h_diagonal_0, sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsUpdateArrayBuffer+marshaledIndex_m_1, h_diagonal_m_1, sizeof(T), hipMemcpyHostToDevice));

    // TODO: time this thing on GPU/CPU? Check this.
    int blockSize = b_dim*stride;
    b_buffer += b_dim*(stride-1);
    topElemBuffer += 1;
    for (int i=0; i<s; i++)
        checkCudaErrors(hipMemcpy(topElemBuffer + i*b_dim, b_buffer + i*blockSize, sizeof(T)*b_dim, hipMemcpyDeviceToDevice));

    b_buffer -= b_dim*(stride-1);
    topElemBuffer -= 1;
    checkCudaErrors(hipMemset(topElemBuffer, 0, sizeof(T)));

    for (int i=0; i<s; i++)
        checkCudaErrors(hipMemcpy(bottomElemBuffer + i*b_dim, b_buffer + i*blockSize, sizeof(T)*b_dim, hipMemcpyDeviceToDevice));

    bottomElemBuffer += 1;
    checkCudaErrors(hipMemset(bottomElemBuffer + s*b_dim - 1, 0, sizeof(T)));

    // finds new RHS with rhsUpdateArrayBuffer having its 1st and last elements modified
    multiply_kernel<T><<<s, b_dim>>>(rhsUpdateArrayBuffer, topElemBuffer, bottomElemBuffer, b_buffer, bNew_buffer, stride, tile);

    // do back data marshaling only in the last step
    if(data->step == (data->totalSteps)-1){
    back_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(b, b_buffer, stride, b_dim, m);
    back_marshaling_bxb<T><<<gridDim, blockDim, marshaling_share_size>>>(bNew, bNew_buffer, stride, b_dim, m);
    // may not be reqd.! check!!!
    }
    
    // updating A in Ax=B. This will be used again in the next step for solving. 
    else{
    checkCudaErrors(hipMemcpy(b_buffer, bNew_buffer, sizeof(T)*data->mPad, hipMemcpyDeviceToDevice));
    
    // modifying main diagonal
    *h_diagonal_0    = cuFma(*h_gammaLeft, *dx_2InvComplex, *(data->constLhsTop));
    *h_diagonal_m_1  = cuFma(*h_gammaRight, *dx_2InvComplex, *(data->constLhsBot));
    
    checkCudaErrors(hipMemcpy(d_buffer, h_diagonal_0, sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_buffer+marshaledIndex_m_1, h_diagonal_m_1, sizeof(T), hipMemcpyHostToDevice));
    }

    // hipMemcpy(h_gammaLeft, d_gamma)
    // hipMemcpy(h_pivotingData, pivotingData, sizeof(int), hipMemcpyDeviceToHost);
    // printf("No of 1 by 1 pivotings done = %d.\n", *h_pivotingData);
    // printf("Solving done.\n\n");
    // free pivotingData both h and dev
    // use checkCudaErrors for all cudaMallocs
    // TODO: check if 2*2 diagonal pivoting is going on... else remove that part...
}

template <typename T, typename T_REAL> 
void tridiagonalSolverHost(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T* bNew, T* rhsUpdateArray, T* x, const int m)
{
    T* h_gammaLeft      = data->h_gammaLeft;
    T* h_gammaRight     = data->h_gammaRight;
    T* dx_2InvComplex   = data->dx_2InvComplex;     // equals -1/(dx*dx)
    T* dx_2InvComplex_1 = data->dx_2InvComplex_1;   // equals +1/(dx*dx)
    T *gamma = data->gamma;
    T *h_diagonal_m_1  = data->h_diagonal_m_1;
    T *h_diagonal_0  = data->h_diagonal_0;
    T  beta = d[0];
    x[0] = cuDiv(b[0], beta);
    int i;
    for (i=1; i<m; i++){
        gamma[i] = cuDiv(du[i-1], beta);
        beta = cuFma(cuNeg(gamma[i]), dl[i], d[i]);
        x[i] = cuFma(cuNeg(x[i-1]), dl[i], b[i]);
        x[i] = cuDiv(x[i], beta);
    }
    int k;
    for (i=1; i<m; i++){
        k = m-i;
        x[k-1] = cuFma(cuNeg(x[k]), gamma[k], x[k-1]);
    }

    *h_gammaLeft     = cuGet<T>((T_REAL)0.0, (T_REAL)0.0);
    *h_gammaRight    = cuGet<T>((T_REAL)0.0, (T_REAL)0.0);
    *h_diagonal_0    = cuFma(*h_gammaLeft, *dx_2InvComplex_1, *(data->constRhsTop));
    *h_diagonal_m_1  = cuFma(*h_gammaRight, *dx_2InvComplex_1, *(data->constRhsBot));
    rhsUpdateArray[0] = *h_diagonal_0;
    rhsUpdateArray[m-1] = *h_diagonal_m_1;

    bNew[0] = cuAdd(cuMul(rhsUpdateArray[0], x[0]), cuMul(*dx_2InvComplex_1, x[1]));
    bNew[m-1] = cuAdd(cuMul(*dx_2InvComplex_1, x[m-2]), cuMul(rhsUpdateArray[m-1], x[m-1]));
    for (i=1; i<m-1; i++){
        // bNew[i] = dx_2InvComplex_1*x[i-1] + rhsUpdateArray[i]*x[i] + dx_2InvComplex_1*x[i+1];
        bNew[i] = cuMul(*dx_2InvComplex_1, x[i-1]);
        bNew[i] = cuFma(rhsUpdateArray[i], x[i], bNew[i]);
        bNew[i] = cuFma(*dx_2InvComplex_1, x[i+1], bNew[i]);
    }

    if (data->step != (data->totalSteps-1)){
        d[0] = cuFma(*h_gammaLeft, *dx_2InvComplex, *(data->constLhsTop));
        d[m-1] = cuFma(*h_gammaLeft, *dx_2InvComplex, *(data->constLhsBot));
        memcpy(b, bNew, sizeof(T)*m);
    }
}

// template<typename T>
// void set_constants(T *dx_2InvComplex);
// template void set_constants<hipComplex>(hipComplex *);
// template 
// void set_constants<hipDoubleComplex>(hipDoubleComplex *);

template <typename T, typename T_REAL> 
void tridiagonalSolver(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T *bNew, T *rhsUpdateArray, const int m);
/* explicit instanciation */
template void tridiagonalSolver<hipComplex, float>(Datablock<hipComplex, float> *data, const hipComplex* dl, hipComplex* d, const hipComplex* du, hipComplex* b, hipComplex *bNew, hipComplex *rhsUpdateArray, const int m);
template void tridiagonalSolver<hipDoubleComplex, double>(Datablock<hipDoubleComplex, double> *data, const hipDoubleComplex* dl, hipDoubleComplex* d, const hipDoubleComplex* du, hipDoubleComplex* b, hipDoubleComplex *bNew, hipDoubleComplex *rhsUpdateArray, const int m);


template <typename T, typename T_REAL> 
void tridiagonalSolverHost(Datablock<T, T_REAL> *data, const T* dl, T* d, const T* du, T* b, T *bNew, T *rhsUpdateArray, T* x, const int m);
// explicit instanciation
template void tridiagonalSolverHost<hipComplex, float>(Datablock<hipComplex, float> *data, const hipComplex* dl, hipComplex* d, const hipComplex* du, hipComplex* b, hipComplex *bNew, hipComplex *rhsUpdateArray, hipComplex *x, const int m);
template void tridiagonalSolverHost<hipDoubleComplex, double>(Datablock<hipDoubleComplex, double> *data, const hipDoubleComplex* dl, hipDoubleComplex* d, const hipDoubleComplex* du, hipDoubleComplex* b, hipDoubleComplex *bNew, hipDoubleComplex *rhsUpdateArray, hipDoubleComplex *x, const int m);